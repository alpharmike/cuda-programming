#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

__global__ void add_matrices(float* c, float* a, float* b, int m, int n) {
	int blockId = blockIdx.x + blockIdx.y * gridDim.x;
	int threadId = blockId * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;

	if (threadId < m * n) {
		c[threadId] = a[threadId] + b[threadId];
	}
}

int main(int argc, char* argv[]) {
	// Set default values for N and M
	int n = 5;
	int m = 5;

	if (argc >= 3) {
		n = atoi(argv[1]);
		m = atoi(argv[2]);

		if (n <= 0 || m <= 0) {
			printf("Positive values should be provided for M and N");
			return EXIT_FAILURE;
		}

	}

	// Initialize matrix size values
	// declare pointers to vectors in host memory
	float* h_mat_a, * h_mat_b, * h_mat_c;
	int mat_size = m * n;
	int mat_size_bytes = mat_size * sizeof(float);

	// Allocate memory for host side matrices
	h_mat_a = (float*)malloc(mat_size_bytes);
	h_mat_b = (float*)malloc(mat_size_bytes);
	h_mat_c = (float*)malloc(mat_size_bytes);

	// Declare device side matrices
	float* d_mat_a, * d_mat_b, * d_mat_c;
	// Allocate memory for device side matrices
	hipMalloc((void**)&d_mat_a, mat_size_bytes);
	hipMalloc((void**)&d_mat_b, mat_size_bytes);
	hipMalloc((void**)&d_mat_c, mat_size_bytes);

	for (int i = 0; i < mat_size; ++i) {
		h_mat_a[i] = i;
		h_mat_b[i] = 10.0 * i;
	}

	hipMemcpy(d_mat_a, h_mat_a, mat_size_bytes, hipMemcpyHostToDevice);
	hipMemcpy(d_mat_b, h_mat_b, mat_size_bytes, hipMemcpyHostToDevice);

	dim3 block_size(16, 16);
	dim3 num_blocks((n - 1 + block_size.x) / block_size.x,
		(m - 1 + block_size.y) / block_size.y);
	add_matrices << < num_blocks, block_size >> > (d_mat_c, d_mat_a, d_mat_b, m, n);

	hipMemcpy(h_mat_c, d_mat_c, mat_size_bytes, hipMemcpyDeviceToHost);

	if (n <= 10) {
		for (int i = 0; i < mat_size; ++i) {
			if (i != 0 && i % m == 0) {
				printf("\n");
			}
			printf("%f ", h_mat_c[i]);
		}
	}


	hipFree(d_mat_a);
	hipFree(d_mat_b);
	hipFree(d_mat_c);
	free(h_mat_a);
	free(h_mat_b);
	free(h_mat_c);

	return 0;
}