
#include "hip/hip_runtime.h"


#include <stdio.h>

__global__ void unique_index_calc(int* data) {
	int tid = threadIdx.x;
	printf("threadIdx: %d, value: %d\n", tid, data[tid]);
}

__global__ void unique_gid_calc(int* data) {
	int tid = threadIdx.x;
	int block_offset = blockIdx.x * blockDim.x;
	int gid = tid + block_offset;
	printf("threadIdx: %d, gid: %d, value: %d\n", tid, gid, data[gid]);
}

__global__ void unique_gid_calc_2d(int* data) {
	// Note we have only one thread in y-dim for each block
	int tid = threadIdx.x;
	int block_offset = blockIdx.x * blockDim.x;
	int row_offset = gridDim.x * blockDim.x * blockIdx.y;
	int gid = tid + block_offset + row_offset;
	printf("blockIdx.x: %d, blockIdx.y: %d, threadIdx: %d, gid: %d, value: %d\n", blockIdx.x, blockIdx.y, tid, gid, data[gid]);
}

__global__ void unique_gid_calc_2d_extended(int* data) {
	int tid = threadIdx.x + threadIdx.y * blockDim.x;
	int num_threads_per_block = blockDim.x * blockDim.y;
	int block_offset = blockIdx.x * num_threads_per_block;
	int number_threads_per_row = num_threads_per_block * gridDim.x;
	int row_offset = number_threads_per_row * blockIdx.y;
	int gid = tid + block_offset + row_offset;
	printf("blockIdx.x: %d, blockIdx.y: %d, threadIdx: %d, gid: %d, value: %d\n", blockIdx.x, blockIdx.y, tid, gid, data[gid]);
}



//int main() {
//	
//	const int array_size = 16;
//	int host_data[array_size] = { 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15, 16 };
//	int array_byte_size = array_size * sizeof(int);
//	int* device_data;
//	cudaMalloc((void**)&device_data, array_byte_size);
//	cudaMemcpy(device_data, host_data, array_byte_size, cudaMemcpyHostToDevice);
//
//
//	/*dim3 block(8);
//	dim3 grid(1);
//	unique_index_calc << <grid, block >> > (device_data);*/
//
//	/*dim3 block(4);
//	dim3 grid(2);
//	unique_gid_calc << <grid, block >> > (device_data);*/
//
//	/*dim3 block(4, 1);
//	dim3 grid(2, 2);
//	unique_gid_calc_2d << <grid, block >> > (device_data);*/
//
//	dim3 block(2, 2);
//	dim3 grid(2, 2);
//	unique_gid_calc_2d_extended << <grid, block >> > (device_data);
//
//	cudaDeviceSynchronize();
//
//	cudaDeviceReset();
//
//	return 0;
//
//}
