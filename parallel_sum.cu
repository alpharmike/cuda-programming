#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>

#include "common.h"
#include "cuda_common.cuh"

__global__ void sum_array_gpu(int* a, int* b, int* c, int size) {
	int gid = threadIdx.x + blockIdx.x * blockDim.x;

	if (gid < size) {
		c[gid] = a[gid] + b[gid];
	}
}

void sum_array_cpu(int* a, int* b, int* c, int size) {
	for (int i = 0; i < size; ++i) {
		c[i] = a[i] + b[i];
	}
}

//int main() {
//	
//	const int array_size = 10000;
//	int array_byte_size = array_size * sizeof(int);
//	const int block_size = 128;
//
//	int* h_a, *h_b, *h_c, *gpu_results;
//	h_a = (int *) malloc(array_byte_size);
//	h_b = (int *) malloc(array_byte_size);
//	h_c = (int *) malloc(array_byte_size);
//	gpu_results = (int *) malloc(array_byte_size);
//
//	for (int i = 0; i < array_size; ++i) {
//		h_a[i] = i;
//		h_b[i] = 2 * i;
//	}
//
//	sum_array_cpu(h_a, h_b, h_c, array_size);
//
//	int* d_a, * d_b, * d_c;
//
//
//	gpuErrCheck(hipMalloc((void**)&d_a, array_byte_size));
//	gpuErrCheck(hipMalloc((void**)&d_b, array_byte_size));
//	gpuErrCheck(hipMalloc((void**)&d_c, array_byte_size));
//
//	hipMemcpy(d_a, h_a, array_byte_size, hipMemcpyHostToDevice);
//	hipMemcpy(d_b, h_b, array_byte_size, hipMemcpyHostToDevice);
//
//
//	dim3 block(block_size);
//	dim3 grid(array_size / block_size + 1);
//
//	sum_array_gpu << <grid, block >> > (d_a, d_b, d_c, array_size);
//
//	hipDeviceSynchronize();
//
//	hipMemcpy(gpu_results, d_c, array_byte_size, hipMemcpyDeviceToHost);
//
//	int same_res = compare_two_arrays(h_c, gpu_results, array_size);
//
//	if (same_res) {
//		printf("Same Results");
//	} else {
//		printf("Different Results");
//	}
//
//
//	hipFree(d_a);
//	hipFree(d_b);
//	hipFree(d_c);
//	free(h_a);
//	free(h_b);
//	free(h_c);
//	free(gpu_results);
//
//	hipDeviceReset();
//
//	return 0;
//
//}
