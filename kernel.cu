﻿
#include "hip/hip_runtime.h"


#include <stdio.h>

__global__ void kernel_func() {
	printf("gridIdx: %d, gridIdy: %d, gridIdz: %d, blockIdx: %d, blockIdy: %d, blockIdz: %d, threadIdx: %d, threadIdy: %d, threadIdz: %d\n", gridDim.x, gridDim.y, gridDim.z, blockIdx.x, blockIdx.y, blockIdx.z, threadIdx.x, threadIdx.y, threadIdx.z);
}

//int main() {
//	dim3 grid(2, 2, 1);
//	dim3 block(4, 2, 1);
//
//	kernel_func <<<grid, block>>>();
//
//	cudaDeviceSynchronize();
//
//	cudaDeviceReset();
//
//	return 0;
//
//}
