
#include "hip/hip_runtime.h"


#include <stdio.h>

__global__ void kernel_exercise() {
	printf("gridX: %d, gridY: %d, gridZ: %d, blockIdx: %d, blockIdy: %d, blockIdz: %d, threadIdx: %d, threadIdy: %d, threadIdz: %d\n", gridDim.x, gridDim.y, gridDim.z, blockIdx.x, blockIdx.y, blockIdx.z, threadIdx.x, threadIdx.y, threadIdx.z);
}

//int main() {
//	int nx = 4;
//	int ny = 4;
//	int nz = 4;
//
//	dim3 block(2, 2, 2);
//	dim3 grid(nx / 2, ny / 2, nz / 2);
//
//	kernel_exercise << <grid, block >> > ();
//
//	cudaDeviceSynchronize();
//
//	cudaDeviceReset();
//
//	return 0;
//
//}
