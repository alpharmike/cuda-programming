
#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>

__global__ void unique_gid_calc_3d(int* data) {
	
	int blockId = blockIdx.x + blockIdx.y * gridDim.x + blockIdx.z * gridDim.x * gridDim.y;
	int globalThreadId = blockId * (blockDim.x * blockDim.y * blockDim.z) + threadIdx.y * blockDim.x + threadIdx.z * (blockDim.x * blockDim.y) + threadIdx.x;

	printf("blockIdx.x: %d, blockIdx.y: %d, blockIdx.z, gid: %d, value: %d\n", blockIdx.x, blockIdx.y, blockIdx.z, globalThreadId, data[globalThreadId]);
}



//int main() {
//	
//	const int array_size = 64;
//	int array_byte_size = array_size * sizeof(int);
//	int* h_data = (int *) malloc(array_byte_size);
//
//	for (int i = 0; i < array_size; ++i) {
//		h_data[i] = i;
//	}
//
//	int* d_data;
//	cudaMalloc((void**)&d_data, array_byte_size);
//	cudaMemcpy(d_data, h_data, array_byte_size, cudaMemcpyHostToDevice);
//
//
//
//	dim3 block(2, 2, 2);
//	dim3 grid(2, 2, 2);
//	unique_gid_calc_3d << <grid, block >> > (d_data);
//
//	cudaDeviceSynchronize();
//
//	cudaFree(d_data);
//	free(h_data);
//
//	cudaDeviceReset();
//
//	return 0;
//
//}
